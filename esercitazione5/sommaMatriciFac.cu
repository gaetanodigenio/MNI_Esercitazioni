#include <assert.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <time.h>

__global__ void sommaGPU(double *a, double *b, double *c, int y);
void sommaCPU(double *a, double *b, double *o, int x, int y);

int main(void){
    //variabili host
    double *a, *b, *c, *o;
    //variabili device
    double *a_d, *b_d, *c_d;
    int x, y;
    hipEvent_t start, stop;

    printf("Inserire dimensione x matrice: \n");
    scanf("%d", &x);

    printf("Inserire dimensione y matrice: \n");
    scanf("%d", &y);

    //configurazione kernel
    dim3 blockDim(8, 8); //ottimale
    dim3 gridDim(x/blockDim.x + (x%blockDim.x == 0 ? 0 : 1), y/blockDim.y + (y%blockDim.y == 0 ? 0 : 1));
    printf("blockDim = (%d,%d)\n", blockDim.x, blockDim.y);
    printf("gridDim = (%d,%d)\n", gridDim.x, gridDim.y);

    //allocazione variabili host
    a = (double*) malloc(x * y * sizeof(double));
    b = (double*) malloc(x * y * sizeof(double));
    c = (double*) malloc(x * y * sizeof(double));
    o = (double*) malloc(x * y * sizeof(double));

    //inizializzazione matrici host
    srand(time(NULL));
    for(int i = 0; i<x; i++){
        for(int j = 0; j<y; j++){
            a[i * y + j] = rand() % 5 - 2;
            b[i * y + j] = rand() % 5 - 2;
        }
    }

    //allocazione variabili device
    hipMalloc((void **) &a_d, (x * y) * sizeof(double));
    hipMalloc((void **) &b_d, (x * y) * sizeof(double));
    hipMalloc((void **) &c_d, (x * y) * sizeof(double));

    //copia host device matrici
    hipMemcpy(a_d, a, x * y * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(b_d, b, x * y * sizeof(double), hipMemcpyHostToDevice);

    //invoco funzione CPU       
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    sommaCPU(a, b, o, x, y);
    hipEventRecord(stop); 
    hipEventSynchronize(stop);
    float elapsed_time1;
    hipEventElapsedTime(&elapsed_time1, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    printf("Tempo CPU: %1.3f\n", elapsed_time1);
    

    //chiamata funzione GPU
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    sommaGPU<<<gridDim, blockDim>>>(a_d, b_d, c_d, y);
    hipEventRecord(stop); 
    hipEventSynchronize(stop);
    float elapsed_time;
    hipEventElapsedTime(&elapsed_time, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    printf("Tempo GPU: %1.3f\n", elapsed_time);

    //copio device host risultato vettore
    hipMemcpy(c, c_d, (x * y) * sizeof(double), hipMemcpyDeviceToHost);

    //stampo risultati
    if (x <= 5 && y <= 5) {
        printf("\nMatrice A:\n");
        for (int i = 0; i < x; i++) {
            for (int j = 0; j < y; j++) {
                printf("%f\t", a[i * y + j]);
            }
            printf("\n");
        }
        printf("\nMatrice B:\n");
        for (int i = 0; i < x; i++) {
            for (int j = 0; j < y; j++) {
                printf("%f\t", b[i * y + j]);
            }
            printf("\n");
        }
        printf("\nMatrice C:\n");
        for (int i = 0; i < x; i++) {
            for (int j = 0; j < y; j++) {
                printf("%f\t", c[i * y + j]);
            }
            printf("\n");
        }
        printf("\nOracolo:\n");
        for (int i = 0; i < x; i++) {
            for (int j = 0; j < y; j++) {
                printf("%f\t", o[i * y + j]);
            }
            printf("\n");
        }
    }

    free(a);
    free(b);
    free(c);
    free(o);
    hipFree(a_d);
    hipFree(b_d);
    hipFree(c_d);
    return 0;
}


__global__ void sommaGPU(double *a, double *b, double *c, int y){
    int i = threadIdx.x + (blockDim.x * blockIdx.x);
    int j = threadIdx.y + (blockDim.y * blockIdx.y);
    c[i * y + j] = a[i * y + j] + b[i * y + j];
}

void sommaCPU(double *a, double *b, double *o, int x, int y){
    for(int i = 0; i<x; i++){
        for(int j = 0; j<y; j++){
            o[i * y + j] = a[i * y + j] + b[i * y + j];
        }
    }
}