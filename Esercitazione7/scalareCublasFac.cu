#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

void scalareCPU(float *a, float *b, float *oracolo, int x, int y);


int main(void){
    hipError_t cudaStat;
    hipblasStatus_t stat;
    hipblasHandle_t handle;
    float *a, *b, *c, *oracolo = 0;
    float *a_d, *b_d, *c_d;
    hipEvent_t start, stop;
    int x, y;

    printf("Inserisci dimensione x matrice: ");
    scanf("%d", &x);

    printf("Inserisci dimensione y matrice: ");
    scanf("%d", &y);

    //allocazione variabili host
    a = (float*)malloc(x * y * sizeof(*a));
    if(!a){
        printf("Errore allocazione a\n");
        return EXIT_FAILURE;
    }

    b = (float*)malloc(y * sizeof(*b));
    if(!b){
        printf("Errore allocazione b\n");
        return EXIT_FAILURE;
    }

    c = (float*)malloc(x * sizeof(*c));
    if(!c){
        printf("Errore allocazione c\n");
        return EXIT_FAILURE;
    }

    oracolo = (float*)malloc(x * sizeof(*oracolo));
    if(!oracolo){
        printf("Errore allocazione oracolo\n");
        return EXIT_FAILURE;
    }

    //inizializzazione matrice e vettore
    srand(time(NULL));
    for(int i = 0; i<x; i++){
        for(int j = 0; j<y; j++){
            a[i * y + j] = rand() % 5 - 2;
        }
    }

    for(int i = 0; i<x; i++){
        b[i] = rand() % 5 - 2;
    }

    //allocazioni variabili device
    cudaStat = hipMalloc ((void**)&a_d, x * y * sizeof(*a));   
    if (cudaStat != hipSuccess) {
        printf ("device memory allocation failed");
        return EXIT_FAILURE;
    }
    
    cudaStat = hipMalloc ((void**)&b_d, y * sizeof(*b));  
    if (cudaStat != hipSuccess) {
        printf ("device memory allocation failed");
        return EXIT_FAILURE;
    }

    cudaStat = hipMalloc ((void**)&c_d, x * sizeof(*c));  
    if (cudaStat != hipSuccess) {
        printf ("device memory allocation failed");
        return EXIT_FAILURE;
    }

    //creazione handle per cublas
    stat = hipblasCreate(&handle);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("CUBLAS initialization failed\n");
        return EXIT_FAILURE;
    }

    stat = hipblasSetMatrix(x, y, sizeof(float), a, x, a_d, x); // Setto h_a su d_a
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("data download failed matrix");
        hipFree (a_d);
        hipblasDestroy(handle);
        return EXIT_FAILURE;
    }
    
    stat = hipblasSetVector(y, sizeof(float), b, 1, b_d, 1);    // Setto h_b su d_b
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("data download failed vector");
        hipFree (b_d);
        hipblasDestroy(handle);
        return EXIT_FAILURE;
    }

    //prodotto matrice vettore cublas
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    float alpha = 1.0;
    float beta = 1.0;
    stat = hipblasSgemv(handle, HIPBLAS_OP_T, x, y, &alpha, a_d, x, b_d, 1, &beta, c_d, 1); //trasposta op_t scalare
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf(" ");
        hipFree (a_d);
        hipFree (b_d);
        hipFree (c_d);
        hipblasDestroy(handle);
        return EXIT_FAILURE;
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float elapsed_time;
    hipEventElapsedTime(&elapsed_time, start, stop);
    printf("\nTempo cublas: %1.3f", elapsed_time);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    //copia risultato device host
    stat = hipblasGetVector(x, sizeof(float), c_d, 1, c, 1); 

    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    scalareCPU(a, b, oracolo, x, y);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float elapsed_time1;
    hipEventElapsedTime(&elapsed_time1, start, stop);
    printf("\nTempo CPU: %1.3f", elapsed_time1);
    hipEventDestroy(start);
    hipEventDestroy(stop);


    //stampo risultati
    if (x <= 10 && y <= 10) {
        printf("\nMatrice A:\n");
        for (int i = 0; i < x; i++) {
            for (int j = 0; j < y; j++) {
                printf("%f\t", a[i * y + j]);
            }
            printf("\n");
        }

        printf("\nVettore B:\n");
        for (int i = 0; i < y; i++) {
            printf("%f\n", b[i]);
        }
            
        printf("\nVettore Prodotto AxB:\n");
        for (int i = 0; i < x; i++) {
            printf("%f\n", c[i]);
        }

        printf("\nOracolo:\n");
        for (int i = 0; i < x; i++) {
            printf("%f\n", oracolo[i]);
        }
    }


    hipFree (a_d);         
    hipFree (b_d);         
    hipFree (c_d);    
    
    hipblasDestroy(handle);  
    
    free(a);     
    free(b);      
    free(c); 
    free(oracolo); 
    return EXIT_SUCCESS;
}

void scalareCPU(float *a, float *b, float *oracolo, int x, int y) {
    for (int i = 0; i < x; i++) {
        oracolo[i] = 0;
        for (int j = 0; j < y; j++) {
            oracolo[i] += a[i * y + j] * b[j];
        }
    }
}