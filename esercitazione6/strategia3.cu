#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>

__global__ void prodottoGPU(float *a, float *b, float *c, int n);
void prodottoCPU(float *a, float *b, float *oracolo, int n);

int main(void){
    //host
    float *a, *b, *c, oracolo = 0.0, somma_totale = 0.0;
    //device
    float *a_d, *b_d, *c_d;
    int n;
    hipEvent_t start, stop;

    printf("Inserisci dimensione vettori: \n");
    scanf("%d", &n);

    //configurazione kernel
    dim3 blockDim(64);
    dim3 gridDim = n / blockDim.x + (n%blockDim.x == 0 ? 0 : 1);
    printf("blockDim = %d\n", blockDim.x);
    printf("gridDim = %d\n", gridDim.x);

    //allocazione variabili host
    a = (float*)malloc(n * sizeof(float));
    b = (float*)malloc(n * sizeof(float));
    c = (float*)malloc(gridDim.x * sizeof(float));


    //inizializzazione variabili host
    srand(time(NULL));
    for(int i = 0; i<n; i++){
        a[i] = rand() % 5 - 2;
        b[i] = rand() % 5 - 2;
    }

    //allocazione variabili device
    hipMalloc((void**) &a_d, n * sizeof(float));
    hipMalloc((void**) &b_d, n * sizeof(float));
    hipMalloc((void**) &c_d, gridDim.x * sizeof(float));

    //passaggio dati host -> device
    hipMemcpy(a_d, a, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(b_d, b, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(c_d, c, gridDim.x * sizeof(float), hipMemcpyHostToDevice);

    //invocazione funzione GPU e copia risultato device -> host
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    prodottoGPU<<<gridDim, blockDim, blockDim.x>>>(a_d, b_d, c_d, n);

    hipMemcpy(c, c_d, gridDim.x * sizeof(float), hipMemcpyDeviceToHost);

    //somma host
    for(int i = 0; i<gridDim.x; i++){
        somma_totale += c[i];
    }
    
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float elapsed_time;
    hipEventElapsedTime(&elapsed_time, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    printf("\nTempo GPU: %1.3f\n", elapsed_time);

    

    //invocazione funzione CPU
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    prodottoCPU(a, b, &oracolo, n);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float elapsed_time1;
    hipEventElapsedTime(&elapsed_time1, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    printf("\nTempo CPU: %1.3f\n", elapsed_time1);

    //print dei risultati
    if (n <= 10) {
        printf("\nA: ");
        for (int i = 0; i < n; i++) {
            printf("%1.3f\t", a[i]);
        }
        printf("\nB: ");
        for (int i = 0; i < n; i++) {
            printf("%1.3f\t", b[i]);
        }
        printf("\nC: ");
        for (int i = 0; i < gridDim.x; i++) {
            printf("%1.3f\t", c[i]);
        }
    }
    printf("\nOracolo: %f\n", oracolo);
    printf("Somma totale host: %1.3f\n", somma_totale);

    free(a);
    free(b);
    free(c);
    hipFree(a_d);
    hipFree(b_d);
    hipFree(c_d);
    return 0;
}

__global__ void prodottoGPU(float *a, float *b, float *c, int n){
    extern __shared__ float v[];
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int id = threadIdx.x;

    if(index < n){
        v[id] = a[index] * b[index];

        __syncthreads();

        //somma parallela
        for(int dist = blockDim.x; dist > 1;){
            dist = dist / 2;
            if(id < dist){
                v[id] = v[id] + v[id + dist];
            }
            __syncthreads();
        }

        if(id == 0){
            //thread 0 ha la somma finale dei prodotti effettuati dai thread del blocco
            c[blockIdx.x] = v[0];
        }

    }
}


void prodottoCPU(float *a, float *b, float *oracolo, int n){
    for(int i = 0; i<n; i++){
        *oracolo += a[i] * b[i];
    }
}