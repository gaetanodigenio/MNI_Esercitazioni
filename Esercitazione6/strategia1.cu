#include <assert.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <time.h>

__global__ void prodottoGPU(double* a, double* b, double* c, int n);
void prodottoCPU(double* a, double *b, double *oracolo, int n);

int main(void){
    //host
    double *a, *b, *c, oracolo = 0.0, somma_totale = 0.0;
    //device
    double *a_d, *b_d, *c_d;
    int n;
    hipEvent_t start, stop; 

    printf("Inserisci numero elementi vettori: \n");
    scanf("%d", &n);

    dim3 blockDim(64);
    dim3 gridDim = n/blockDim.x + ((n % blockDim.x) == 0 ? 0 : 1);
    printf("gridDim: %d\n", gridDim.x);
    printf("BlockDim: %d\n", blockDim.x);

    //allocazione variabili host
    a = (double*)malloc(n * sizeof(double));
    b = (double*)malloc(n * sizeof(double));
    c = (double*)malloc(n * sizeof(double));

    //allocazione variabili device
    hipMalloc((void**) &a_d, n * sizeof(double));
    hipMalloc((void**) &b_d, n * sizeof(double));
    hipMalloc((void**) &c_d, n * sizeof(double));

    //inizializzazione variabili host
    srand((unsigned int) time(0)); 
    for(int i = 0; i<n; i++){
        a[i] = rand() % 5 - 2;
        b[i] = rand() % 5 - 2;
    }

   
    //copia variabili host -> device
    hipMemcpy(a_d, a, n*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(b_d, b, n*sizeof(double), hipMemcpyHostToDevice);
    memset(c, 0, n * sizeof(double));
	hipMemset(c_d, 0, n * sizeof(double));

    //funzione su GPU
    hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start); 

    prodottoGPU<<<gridDim, blockDim>>>(a_d, b_d, c_d, n);

    //copia risultato device -> host
    hipMemcpy(c, c_d, n * sizeof(double), hipMemcpyDeviceToHost);

    //somma sequenziale su host
    for(int i = 0; i<n; i++){
        somma_totale += c[i];
    }

    hipEventRecord(stop); 
	hipEventSynchronize(stop);
	float elapsed;  
	hipEventElapsedTime(&elapsed, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	printf("Tempo totale GPU: %1.3f ", elapsed);

    

    //funzione CPU per calcolo speedup e oracolo
    hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);

    prodottoCPU(a, b, &oracolo, n);

    hipEventRecord(stop);
	hipEventSynchronize(stop); 
	float elapsed1;  
	hipEventElapsedTime(&elapsed1, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	printf("Tempo totale CPU: %1.3f ", elapsed1);

    if(n < 20){
        for(int i = 0; i<n; i++){
            printf("\na[%d] = %1.3f ", i, a[i]);
        }
        printf("\n");

        for(int i = 0; i<n; i++){
            printf("\nb[%d] = %1.3f ", i, b[i]);
        }
        printf("\n");

        for(int i = 0; i<n; i++){
            printf("\nc[%d] = %1.3f ", i, c[i]);
        }
        printf("\n");
    }

    printf("Somma totale: %1.3f\n", somma_totale);
    printf("Oracolo: %1.3f\n", oracolo);

    free(a);
    free(b);
    free(c);
    hipFree(a_d);
    hipFree(b_d);
    hipFree(c_d);

    return 0;
}

__global__ void prodottoGPU(double* a, double* b, double* c, int n){
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if(index < n){
        c[index] = a[index] * b[index];
    }
}


void prodottoCPU(double* a, double *b, double *oracolo, int n){
    for(int i = 0; i<n; i++){
        *oracolo += a[i] * b[i];
    }
}
