#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>

__global__ void prodottoGPU(float* a, float* b, float* c, int n);
void prodottoCPU(float* a, float* b, float* oracolo, int n);

int main(void){
    //host
    float *a, *b, *c, oracolo = 0.0, somma_totale = 0.0;
    //device
    float *a_d, *b_d, *c_d;
    int n;
    hipEvent_t start, stop;

    printf("Inserisci dimensione dei vettori: \n");
    scanf("%d", &n);

    //configurazione kernel
    dim3 blockDim(64);
    dim3 gridDim = n/blockDim.x + (n%blockDim.x == 0 ? 0 : 1);
    printf("blockDim = %d\n", blockDim.x);
    printf("gridDim = %d\n", gridDim.x);

    //allocazione host
    a = (float*)malloc(n * sizeof(float));
    b = (float*)malloc(n * sizeof(float));
    c = (float*)malloc(gridDim.x * sizeof(float));

    //inizializzazione host
    srand(time(NULL));
    for (int i = 0; i < n; i++) {
        a[i] = rand() % 5 - 2; 
        b[i] = rand() % 5 - 2;
    }

    //allocazione device
    hipMalloc((void**) &a_d, n * sizeof(float));
    hipMalloc((void**) &b_d, n * sizeof(float));
    hipMalloc((void**) &c_d, gridDim.x * sizeof(float));
    
    //copia dati host -> device
    hipMemcpy(a_d, a, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(b_d, b, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(c_d, c, gridDim.x * sizeof(float), hipMemcpyHostToDevice);

    //invocazione funzione GPU e invio risultato device -> host
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    prodottoGPU<<<gridDim, blockDim, blockDim.x>>>(a_d, b_d, c_d, n);

    hipMemcpy(c, c_d, gridDim.x * sizeof(float), hipMemcpyDeviceToHost);

    /* somma sull'host */
    for (int i = 0; i < gridDim.x; i++) {
        somma_totale += c[i];
    }

    hipEventRecord(stop); 
    hipEventSynchronize(stop);
    float elapsed_time;
    hipEventElapsedTime(&elapsed_time, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    printf("Tempo GPU: %1.3f\n", elapsed_time);

    


    //invocazione funzione CPU per confronto e calcolo tempi
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    prodottoCPU(a, b, &oracolo, n);

    hipEventRecord(stop); 
    hipEventSynchronize(stop);
    float elapsed_time1;
    hipEventElapsedTime(&elapsed_time1, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    printf("Tempo CPU: %1.3f\n", elapsed_time1);

    //stampo risultati
    if(n <= 10){
        printf("\nA: ");
        for (int i = 0; i < n; i++) {
            printf("%1.3f\t", a[i]);
        }
        printf("\nB: ");
        for (int i = 0; i < n; i++) {
            printf("%1.3f\t", b[i]);
        }
        printf("\nC: ");
        for(int i = 0; i<gridDim.x; i++){
            printf("%1.3f\t", c[i]);
        }
    }
    
    printf("Oracolo: %1.3f\n", oracolo);
    printf("Somma su host totale: %1.3f\n", somma_totale);

    free(a);
    free(b);
    free(c);
    hipFree(a_d);
    hipFree(b_d);
    hipFree(c_d);
    return 0;
}

__global__ void prodottoGPU(float* a, float* b, float* c, int n){
    extern __shared__ float v[]; //allocazione dinamica 3 parametro invocazione kernel

    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int id = threadIdx.x;

    if(index < n){
        v[id] = a[index] * b[index];

        __syncthreads();

        //somma parallela
        for(int dist = 1; dist < blockDim.x; dist = dist << 1){
            if((id % (dist * 2)) == 0){
                v[id] = v[id] + v[id + dist];
            }
            __syncthreads();
        }

        if(id == 0){
            c[blockIdx.x] = v[0];
        }

    }

}

void prodottoCPU(float* a, float* b, float* oracolo, int n){
    for(int i = 0; i<n; i++){
        *oracolo += a[i] * b[i];
    }
}
