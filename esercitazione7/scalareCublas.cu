#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#define M 3

int main (void){
    hipError_t cudaStat;
    hipblasStatus_t stat;
    hipblasHandle_t handle;
    float *h_a = 0;     // Host array a
    float *d_a;         // Device array a
    float *h_b = 0;     // Host array b
    float *d_b;         // Device array b
    float result = 0;   // Risultato finale
	
	/*
	[3, 10, 20] * [5, 10, 15] = 415
	*/

    h_a = (float *)malloc (M * sizeof (*h_a));      // Alloco h_a e lo inizializzo
    if (!h_a) {
        printf ("host memory allocation failed");
        return EXIT_FAILURE;
    }
    h_a[0] = 3.0;
    h_a[1] = 10.0;
    h_a[2] = 20.0;
    
    h_b = (float *)malloc (M * sizeof (*h_b));  // Alloco h_b e lo inizializzo
    if (!h_b) {
        printf ("host memory allocation failed");
        return EXIT_FAILURE;
    }
    h_b[0] = 5.0;
    h_b[1] = 10.0;
    h_b[2] = 15.0;
    
    cudaStat = hipMalloc ((void**)&d_a, M*sizeof(*h_a));       // Alloco d_a
    if (cudaStat != hipSuccess) {
        printf ("device memory allocation failed");
        return EXIT_FAILURE;
    }
    
    cudaStat = hipMalloc ((void**)&d_b, M*sizeof(*h_b));       // Alloco d_b
    if (cudaStat != hipSuccess) {
        printf ("device memory allocation failed");
        return EXIT_FAILURE;
    }
    
    stat = hipblasCreate(&handle);               // Creo l'handle per cublas
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("CUBLAS initialization failed\n");
        return EXIT_FAILURE;
    }
    

    stat = hipblasSetVector(M,sizeof(float),h_a,1,d_a,1);    // Setto h_a su d_a
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("data download failed");
        hipFree (d_a);
        hipblasDestroy(handle);
        return EXIT_FAILURE;
    }
    

    stat = hipblasSetVector(M,sizeof(float),h_b,1,d_b,1);    // Setto h_b su d_b
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("data download failed");
        hipFree (d_b);
        hipblasDestroy(handle);
        return EXIT_FAILURE;
    }
    
    ----
    stat = hipblasSdot(handle,M,d_a,1,d_b,1,&result);        // Calcolo il prodotto
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("data download failed hipblasSdot");
        hipFree (d_a);
        hipFree (d_b);
        hipblasDestroy(handle);
        return EXIT_FAILURE;
    }
    
    printf("Risultato del prodotto --> %f",result);
    
    hipFree (d_a);     // Dealloco d_a
    hipFree (d_b);     // Dealloco d_b
    
    hipblasDestroy(handle);  // Distruggo l'handle
    
    free(h_a);      // Dealloco h_a
    free(h_b);      // Dealloco h_b    
    return EXIT_SUCCESS;
}