#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

void scalareCPU(float *a, float *b, float *oracolo, int n);

int main(void){
    //host
    float *a = 0, *b = 0, oracolo = 0.0;
    //device
    float *a_d, *b_d;
    float result = 0;
    hipEvent_t start, stop;
    hipError_t cudaStat;
    hipblasStatus_t stat;
    hipblasHandle_t handle;
    int M;

    printf("Inserisci lunghezza vettori: \n");
    scanf("%d", &M);

    //allocazione variabili host
    a = (float*)malloc(M * sizeof(*a));
    if(!a){
        printf("Allocazione non riuscita\n");
        return EXIT_FAILURE;
    }
    a[0] = 3.0;
    a[1] = 10.0;
    a[2] = 20.0;

    b = (float*)malloc(M * sizeof(*b));
    if(!b){
        printf("Allocazione non riuscita\n");
        return EXIT_FAILURE;
    }
    b[0] = 5.0;
    b[1] = 10.0;
    b[2] = 15.0;

    //allocazione variabili device
    cudaStat = hipMalloc((void**) &a_d, M * sizeof(*a));
    if(cudaStat != hipSuccess){
        printf("Allocazione memoria device non riuscita\n");
        return EXIT_FAILURE;
    }
    
    
    cudaStat = hipMalloc((void**) &b_d, M * sizeof(*b));
    if(cudaStat != hipSuccess){
        printf("Allocazione memoria device non riuscita\n");
        return EXIT_FAILURE;
    }

    //creazione handle per cublas
    stat = hipblasCreate(&handle);
    if(stat != HIPBLAS_STATUS_SUCCESS){
        printf("Inizializzazione cublas errore\n");
        return EXIT_FAILURE;
    }

    //copia a da host a device
    stat = hipblasSetVector(M, sizeof(float), a, 1, a_d, 1);
    if(stat != HIPBLAS_STATUS_SUCCESS){
        printf("Fallimento passaggio dati\n");
        hipFree(a_d);
        hipblasDestroy(handle);
        return EXIT_FAILURE;
    }

    stat = hipblasSetVector(M, sizeof(float), b, 1, b_d, 1);
    if(stat != HIPBLAS_STATUS_SUCCESS){
        printf("Fallimento passagio dati\n");
        hipFree(b_d);
        hipblasDestroy(handle);
        return EXIT_FAILURE;
    }

    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    //calcolo del prodotto
    stat = hipblasSdot(handle, M, a_d, 1, b_d, 1, &result);
    if(stat != HIPBLAS_STATUS_SUCCESS){
        printf("Fallimento calcolo prodotto");
        hipFree(a_d);
        hipFree(b_d);
        hipblasDestroy(handle);
        return EXIT_FAILURE;
    }

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float elapsed_time;
    hipEventElapsedTime(&elapsed_time, start, stop);
    printf("\nTempo cublas: %1.3f", elapsed_time);


    hipFree(a_d);
    hipFree(b_d);
    
    hipblasDestroy(handle);

    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    scalareCPU(a, b, &oracolo, M);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float elapsed_time1;
    hipEventElapsedTime(&elapsed_time1, start, stop);
    printf("\nTempo CPU: %1.3f", elapsed_time1);

    free(a);
    free(b);
    return EXIT_SUCCESS;
}

void scalareCPU(float *a, float *b, float *oracolo, int n){
    for(int i = 0; i<n; i++){
        *oracolo += a[i] * b[i];
    }
}