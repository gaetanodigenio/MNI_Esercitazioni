/*
Cognome e nome: Di Genio Gaetano
Matricola: 0522501339
*/

#include <assert.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <time.h>

__global__ void sommaMatriciGPU(double *a, double *b, double *s, int n);
void sommaMatriciCPU(double *a, double *b, double *s, int n);

int main(void){
    //host
    double *a_h, *b_h, *s_h, *s_h2;
    //device
    double *a_d, *b_d, *s_d;
    int n;

    printf("Inserisci dimensione matrice: \n");
    scanf("%d", &n);

    //configurazione kernel
    dim3 blockDim(16, 16);
    dim3 gridDim(n/blockDim.x + ((n%blockDim.x)==0?0:1), n/blockDim.y + ((n%blockDim.y)==0?0:1));

    //allocazione variabili host
    a_h = (double*) malloc(n * n * sizeof(double));
    b_h = (double*) malloc(n * n * sizeof(double));
    s_h = (double*) malloc(n * n * sizeof(double));
    s_h2 = (double*) malloc(n * n * sizeof(double));

    //inizializzazione matrici host
    srand(time(NULL));
    for(int i = 0; i<n; i++){
        for(int j = 0; j<n; j++){
            a_h[i * n + j] = rand() % 5 - 2;
            b_h[i * n + j] = rand() % 5 - 2;
        }
    }

    //allocazione variabili device
    hipMalloc((void**) &a_d, n * n * sizeof(double));
    hipMalloc((void**) &b_d, n * n * sizeof(double));
    hipMalloc((void**) &s_d, n * n * sizeof(double));

    //copia da host a device dei vettori
    hipMemcpy(a_d, a_h, n * n * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(b_d, b_h, n * n * sizeof(double), hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    //invoco kernel
    sommaMatriciGPU<<<blockDim, gridDim>>>(a_d, b_d, s_d, n);
    //copio risultato da device a host
    hipMemcpy(s_h, s_d, n * n * sizeof(double), hipMemcpyDeviceToHost);
    hipEventRecord(stop); 
    hipEventSynchronize(stop);
    float elapsed_time;
    hipEventElapsedTime(&elapsed_time, start, stop);
    if (n <= 10) {
        printf("\nMatrice A:\n");
        for (int i = 0; i < n; i++) {
            for (int j = 0; j < n; j++) {
                printf("%f\t", a_h[i * n + j]);
            }
            printf("\n");
        }
        printf("\nMatrice B:\n");
        for (int i = 0; i < n; i++) {
            for (int j = 0; j < n; j++) {
                printf("%f\t", b_h[i * n + j]);
            }
            printf("\n");
        }
        printf("\nMatrice Somma:\n");
        for (int i = 0; i < n; i++) {
            for (int j = 0; j < n; j++) {
                printf("%f\t", s_h[i * n + j]);
            }
            printf("\n");
        }
    }
    printf("\n\nTempo di esecuzione GPU: %f\n", elapsed_time);

    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    //invoco funzione CPU
    sommaMatriciCPU(a_h, b_h, s_h2, n);
    
    //copio risultato da device a host
    hipEventRecord(stop); 
    hipEventSynchronize(stop);
    float elapsed_time1;
    hipEventElapsedTime(&elapsed_time1, start, stop);
    printf("\n\nTempo di esecuzione CPU: %f\n", elapsed_time1);

    free(a_h);
    free(b_h);
    free(s_h);
    hipFree(a_d);
    hipFree(b_d);
    hipFree(s_d);
    return 0;
}

__global__ void sommaMatriciGPU(double *a, double *b, double *s, int n){
    int i = threadIdx.x + (blockDim.x * blockIdx.x);
    int j = threadIdx.y + (blockDim.y * blockIdx.y);
    s[i * n + j] = a[i * n + j] + b[i * n + j];
}

void sommaMatriciCPU(double *a, double *b, double *s, int n){
    for(int i = 0; i<n; i++){
        for(int j= 0; j<n; j++){
            s[i * n + j] = a[i * n + j] + b[i * n +j];
        }
    }

}