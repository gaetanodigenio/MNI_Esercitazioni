/*
Cognome e nome: Di Genio Gaetano
Matricola: 0522501339
*/

#include <assert.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <time.h>

void prodottoCPU(double *a, double *b, double *c, int n);
__global__ void prodottoGPU(double* a, double* b, double* c, int n);

int main(void)
{
 double *a_h, *b_h, *c_h, c_h2 = 0.0; //host data
 double *a_d, *b_d, *c_d; // device data
 int N, nBytes, i ;
dim3 gridDim, blockDim;


printf("***\t PRODOTTO DI DUE VETTORI \t***\n");
printf("Inserisci il numero degli elementi dei vettori\n");
scanf("%d",&N); 

blockDim.x = 64;

//determinazione esatta del numero di blocchi
gridDim = N/blockDim.x + ((N%blockDim.x)==0?0:1);

printf("\nBlockDim: %d\n", blockDim.x);
printf("gridDim: %d\n", gridDim.x);

 nBytes = N*sizeof(float);
 a_h = (double *)malloc(nBytes);
 b_h = (double *)malloc(nBytes);
 c_h = (double *)malloc(nBytes);
 hipMalloc((void **) &a_d, nBytes);
 hipMalloc((void **) &b_d, nBytes);
 hipMalloc((void **) &c_d, nBytes);
 // inizializzo i dati
 /*Inizializza la generazione random dei vettori utilizzando l'ora attuale del sistema*/                
srand((unsigned int) time(0)); 
        
 for (i=0; i<N; i++) {
	a_h[i] = rand()%5-2;
	b_h[i] = rand()%5-2;;
 }
  		
 hipMemcpy(a_d, a_h, nBytes, hipMemcpyHostToDevice);
 hipMemcpy(b_d, b_h, nBytes, hipMemcpyHostToDevice);
 

 //azzeriamo il contenuto del vettore c
memset(c_h, 0, nBytes);
hipMemset(c_d, 0, nBytes);

//tempo esecuzione parallelo
hipEvent_t start, stop; 
hipEventCreate(&start);
hipEventCreate(&stop);
hipEventRecord(start); 

prodottoGPU<<<gridDim, blockDim>>>(a_d, b_d, c_d, N);

hipEventRecord(stop); 
hipEventSynchronize(stop);
float elapsed;  
hipEventElapsedTime(&elapsed, start, stop);
hipEventDestroy(start);
hipEventDestroy(stop);

printf("Tempo totale GPU: %1.3f ", elapsed);

hipMemcpy(c_h, c_d, nBytes, hipMemcpyDeviceToHost);


 
 // calcolo prodotto seriale su CPU e tempi
hipEventCreate(&start);
hipEventCreate(&stop);
hipEventRecord(start);

prodottoCPU(a_h, b_h, &c_h2, N);

hipEventRecord(stop);
hipEventSynchronize(stop); 
float elapsed1;  
hipEventElapsedTime(&elapsed1, start, stop);
hipEventDestroy(start);
hipEventDestroy(stop);
printf("Tempo totale CPU: %1.3f ", elapsed1);

 
 
 if (N<20){
 	for(i=0;i<N;i++)
		printf("\na_h[%d]=%6.2f ",i, a_h[i]);
	printf("\n");

	for(i=0;i<N;i++)
		printf("\nb_h[%d]=%6.2f ",i, b_h[i]);
	printf("\n");

	for(i=0;i<N;i++)
		printf("\nc_h[%d]=%6.2f ",i, c_h[i]);
	printf("\n");
}


 free(a_h); 
 free(b_h); 
 free(c_h); 
 hipFree(a_d); 
 hipFree(b_d); 
 hipFree(c_d);

 return 0;
}

//Seriale
void prodottoCPU(double *a, double *b, double *c, int n)
{
    for(int i=0;i<n;i++)
        *c +=a[i] * b[i];
} 

//Parallelo
__global__ void prodottoGPU(double* a, double * b, double* c, int n){
    int index=threadIdx.x + blockIdx.x*blockDim.x;
    if(index < n)
        c[index] = a[index] * b[index];
}
